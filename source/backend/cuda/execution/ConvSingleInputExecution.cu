#include "hip/hip_runtime.h"
//
//  ConvSingleInputExecution.cpp
//  MNN
//
//  Created by MNN on 2020/08/22.
//  Copyright © 2018, Alibaba Group Holding Limited
//

#include "ConvSingleInputExecution.hpp"

namespace MNN {
namespace CUDA {

template <typename T>
__global__ void Pad(const size_t size, const T* input, const int old_height,
                    const int old_width, const int padded_height, const int padded_width, const int pad_top,
                    const int pad_left, float pad_value, T* output) {
    T pad_value_ = static_cast<T>(pad_value);
    for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
        int block_num = pos / (padded_width*padded_height);
        int left = pos % (padded_width*padded_height);
        const int padded_w = left % padded_width;
        const int padded_h = left / padded_width % padded_height;
        if (padded_h - pad_top < 0 || padded_w - pad_left < 0 || padded_h - pad_top >= old_height ||
              padded_w - pad_left >= old_width) {
            output[pos] = pad_value_;
        } else {
            output[pos] = input[(block_num * old_height + padded_h - pad_top) * old_width + padded_w - pad_left];
        }
    }
    return;
}

ConvSingleInputExecution::Resource::Resource(Backend* bn, const MNN::Op* op) {
    mBackend = bn;
    auto conv       = op->main_as_Convolution2D();
    auto common     = conv->common();
    cudnn_data_type_ = HIPDNN_DATA_FLOAT;
    cudnn_data_type_len_ = 0;
    mKernelInfo.kernelX        = common->kernelX();
    mKernelInfo.kernelY        = common->kernelY();
    mKernelInfo.groups         = common->group();
    mKernelInfo.padMode        = common->padMode();
    mKernelInfo.padX           = common->padX();
    mKernelInfo.padY           = common->padY();

    if (nullptr != common->pads()) {
        mKernelInfo.padX = common->pads()->data()[1];
        mKernelInfo.padY = common->pads()->data()[0];
    }
    mKernelInfo.strideX        = common->strideX();
    mKernelInfo.strideY        = common->strideY();
    mKernelInfo.dilateX        = common->dilateX();
    mKernelInfo.dilateY        = common->dilateY();
    mKernelInfo.activationType = common->relu() ? 1 : (common->relu6() ? 2 : 0);
    use_relu_ = (mKernelInfo.activationType == 1);
    use_relu6_ = (mKernelInfo.activationType == 2);
    use_bias_ = true;
    cudnn_check(hipdnnCreateActivationDescriptor(&act_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&bias_desc_));
    cudnn_check(hipdnnCreateFilterDescriptor(&filter_desc_));
    cudnn_check(hipdnnCreateConvolutionDescriptor(&conv_desc_));

    //weight host->device
    const float* filterDataPtr = nullptr;
    int weightSize = 0;
    std::shared_ptr<ConvolutionCommon::Int8Common> quanCommon;
    ConvolutionCommon::getConvParameters(&quanCommon, conv, &filterDataPtr, &weightSize);
    mKernelInfo.kernelN = common->outputCount();
    mKernelInfo.kernelC = weightSize / mKernelInfo.kernelN / mKernelInfo.kernelX / mKernelInfo.kernelY;

    weightTensor.reset(Tensor::createDevice<float>({weightSize}));
    bn->onAcquireBuffer(weightTensor.get(), Backend::STATIC);
    mFilter = (void *)weightTensor.get()->buffer().device;
    cuda_check(hipMemcpy(mFilter, filterDataPtr, weightSize*sizeof(float), hipMemcpyHostToDevice));

    int biasSize = conv->bias()->size();
    biasTensor.reset(Tensor::createDevice<float>({biasSize}));
    bn->onAcquireBuffer(biasTensor.get(), Backend::STATIC);
    mBias = (void *)biasTensor.get()->buffer().device;

    cuda_check(hipMemcpy(mBias, conv->bias()->data(), conv->bias()->size()*sizeof(float), hipMemcpyHostToDevice));

    int bias_size = conv->bias()->size();
    int dim_bias[] = {1, bias_size, 1, 1};
    int stride_bias[] = {bias_size, 1, 1, 1};
    if(cudnn_data_type_ == HIPDNN_DATA_FLOAT) {
        cudnn_check(hipdnnSetTensorNdDescriptor(bias_desc_, HIPDNN_DATA_FLOAT, 4, dim_bias, stride_bias));
    }
    else if(cudnn_data_type_ == HIPDNN_DATA_HALF) {
        cudnn_check(hipdnnSetTensorNdDescriptor(bias_desc_, HIPDNN_DATA_HALF, 4, dim_bias, stride_bias));
    } else {
        MNN_PRINT("only supports fp32/fp16 data type!!!\n");
    }
    use_bias_ = true;

    mKernelInfo.kernelN = common->outputCount();
    mKernelInfo.kernelC = weightSize / (mKernelInfo.kernelN * mKernelInfo.kernelY * mKernelInfo.kernelX);
    std::vector<int> filter_shape = {mKernelInfo.kernelN, mKernelInfo.kernelC, mKernelInfo.kernelY, mKernelInfo.kernelX};

    cudnn_check(hipdnnSetFilter4dDescriptor(filter_desc_, cudnn_data_type_, HIPDNN_TENSOR_NCHW, filter_shape[0],
        filter_shape[1], filter_shape[2], filter_shape[3]));
    cudnn_check(hipdnnSetConvolution2dDescriptor(conv_desc_, 0, 0, mKernelInfo.strideY, mKernelInfo.strideX, 
            mKernelInfo.dilateY, mKernelInfo.dilateX, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    if (cudnn_data_type_ == HIPDNN_DATA_HALF) {
        cudnn_check(hipdnnSetConvolutionMathType(conv_desc_, HIPDNN_TENSOR_OP_MATH));
    }
    //set group num
    cudnn_check(hipdnnSetConvolutionGroupCount(conv_desc_, mKernelInfo.groups));
    if(use_relu_) {
        cudnn_check(hipdnnSetActivationDescriptor(act_desc_, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0));
    } else if(use_relu6_) {
        cudnn_check(hipdnnSetActivationDescriptor(act_desc_, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, 6.0));
    } else {
        //do nothing
    }
}

ConvSingleInputExecution::Resource::~Resource() {
    cudnn_check(hipdnnDestroyFilterDescriptor(filter_desc_));
    cudnn_check(hipdnnDestroyTensorDescriptor(bias_desc_));
    cudnn_check(hipdnnDestroyActivationDescriptor(act_desc_));
    cudnn_check(hipdnnDestroyConvolutionDescriptor(conv_desc_));
}

ConvSingleInputExecution::ConvSingleInputExecution(Backend* backend, const MNN::Op* op) : Execution(backend), mOp(op) {
    //MNN_PRINT("cuda convSingleInput onInit in\n");
    mResource.reset(new Resource(backend, op));
    cudnn_handle_ = nullptr;
    input_desc_ = nullptr;
    output_desc_ = nullptr;
    padded_desc_ = nullptr;
    auto runtime = static_cast<CUDABackend*>(backend)->getCUDARuntime();
    cudnn_handle_ = runtime->cudnn_handle();
    cudnn_check(hipdnnCreateTensorDescriptor(&input_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&output_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&padded_desc_));
}
ConvSingleInputExecution::ConvSingleInputExecution(Backend* backend, const MNN::Op* op, std::shared_ptr<Resource> res) : Execution(backend), mOp(op) {
    mResource = res;
    cudnn_handle_ = nullptr;
    input_desc_ = nullptr;
    output_desc_ = nullptr;
    padded_desc_ = nullptr;
    auto runtime = static_cast<CUDABackend*>(backend)->getCUDARuntime();
    cudnn_handle_ = runtime->cudnn_handle();
    cudnn_check(hipdnnCreateTensorDescriptor(&input_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&output_desc_));
    cudnn_check(hipdnnCreateTensorDescriptor(&padded_desc_));
}

ConvSingleInputExecution::~ConvSingleInputExecution() {
    cudnn_check(hipdnnDestroyTensorDescriptor(padded_desc_));
    cudnn_check(hipdnnDestroyTensorDescriptor(output_desc_));
    cudnn_check(hipdnnDestroyTensorDescriptor(input_desc_));
}
bool ConvSingleInputExecution::onClone(Backend* bn, const Op* op, Execution** dst) {
    if (!mValid) {
        return false;
    }
    if (nullptr == dst) {
        return true;
    }
    auto dstExe = new ConvSingleInputExecution(bn, op, mResource);
    *dst = dstExe;
    return true;
}

ErrorCode ConvSingleInputExecution::onResize(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    // prepare
    //MNN_PRINT("cuda convSingleInput onResize in, pad:%d\n", mKernelInfo.padX);
    auto input = inputs[0], output = outputs[0];

    mIOInfo.iw = input->width();
    mIOInfo.ih = input->height();
    mIOInfo.ic = input->channel();
    mIOInfo.ib = input->batch();
    
    mIOInfo.ow = output->width();
    mIOInfo.oh = output->height();
    mIOInfo.oc = output->channel();
    mIOInfo.ob = output->batch();

    if(mIOInfo.iw==0) {
        mIOInfo.iw = 1;
    }
    if(mIOInfo.ih==0) {
        mIOInfo.ih = 1;
    }
    if(mIOInfo.ic==0) {
        mIOInfo.ic = 1;
    }
    if(mIOInfo.ib==0) {
        mIOInfo.ib = 1;
    }
    if(mIOInfo.ow==0) {
        mIOInfo.ow = 1;
    }
    if(mIOInfo.oh==0) {
        mIOInfo.oh = 1;
    }
    if(mIOInfo.oc==0) {
        mIOInfo.oc = 1;
    }
    if(mIOInfo.ob==0) {
        mIOInfo.ob = 1;
    }
    std::vector<int> in_shape = {mIOInfo.ib, mIOInfo.ic, mIOInfo.ih, mIOInfo.iw};
    std::vector<int> output_shape = {mIOInfo.ob, mIOInfo.oc, mIOInfo.oh, mIOInfo.ow};
    auto cudnn_data_type_ = mResource->cudnn_data_type_;
    auto pads = ConvolutionCommon::convolutionPadFull(input, output, mOp->main_as_Convolution2D()->common());
    pad_left_ = std::get<0>(pads);
    pad_top_ = std::get<1>(pads);
    pad_right_ = std::get<2>(pads);
    pad_bottom_ = std::get<3>(pads);
    // printf("filter:%d %d %d %d\n", filter_shape[0], filter_shape[1], filter_shape[2], filter_shape[3]);
    // printf("input:%d %d %d %d\n", in_shape[0], in_shape[1], in_shape[2], in_shape[3]);
    // printf("output:%d %d %d %d\n", output_shape[0], output_shape[1], output_shape[2], output_shape[3]);
    cudnn_check(hipdnnSetTensor4dDescriptor(input_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, in_shape[0],
                                in_shape[1], in_shape[2], in_shape[3]));

    cudnn_check(hipdnnSetTensor4dDescriptor(output_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, output_shape[0],
                                output_shape[1], output_shape[2], output_shape[3]));

    hipdnnTensorDescriptor_t input_descriptor_real = nullptr;
    use_pad_ = (pad_left_!=0 || pad_right_!=0 || pad_top_!=0 || pad_bottom_!=0 ) ? true : false;

    if(use_pad_) {
        int totalSize = in_shape[0]*in_shape[1]*(in_shape[2]+pad_top_+pad_bottom_)*(in_shape[3]+pad_left_+pad_right_);
        padTensor.reset(Tensor::createDevice<float>({totalSize}));
        backend()->onAcquireBuffer(padTensor.get(), Backend::DYNAMIC);
        mPadPtr = (void *)padTensor.get()->buffer().device;

        //dynamic memory release
        backend()->onReleaseBuffer(padTensor.get(), Backend::DYNAMIC);

        cudnn_check(hipdnnSetTensor4dDescriptor(padded_desc_, HIPDNN_TENSOR_NCHW, cudnn_data_type_, in_shape[0], in_shape[1],
                                in_shape[2] + +pad_top_+pad_bottom_, in_shape[3] + pad_left_+pad_right_));
    }
    input_descriptor_real = use_pad_ ? padded_desc_ : input_desc_;

    // algorithm
    constexpr int requested_algo_count = 1;
    int returned_algo_count;
    hipdnnConvolutionFwdAlgoPerf_t perf_results;
    cudnn_check(cudnnGetConvolutionForwardAlgorithm_v7(cudnn_handle_, input_descriptor_real, mResource->filter_desc_, mResource->conv_desc_,
                                                output_desc_, requested_algo_count, &returned_algo_count, &perf_results));
    conv_algorithm_ = perf_results.algo;
    auto& mKernelInfo = mResource->mKernelInfo;

    if(mIOInfo.iw==1 && mIOInfo.ih==1 && mKernelInfo.kernelY==1 && mKernelInfo.kernelX==1) {
        conv_algorithm_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    }
    // workspace
    cudnn_check(hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle_, input_descriptor_real, mResource->filter_desc_, mResource->conv_desc_, output_desc_,
                                            conv_algorithm_, &workspace_size_));

    if (workspace_size_ != 0) {
        int workspaceSize = workspace_size_;
        workspaceTensor.reset(Tensor::createDevice<float>({workspaceSize}));
        //cudnn not support workspace memory reuse
        backend()->onAcquireBuffer(workspaceTensor.get(), Backend::STATIC);
        mWorkSpace = (void *)workspaceTensor.get()->buffer().device;
    }
    //MNN_PRINT("cuda convSingleInput onResize out\n");
    return NO_ERROR;
}

ErrorCode ConvSingleInputExecution::onExecute(const std::vector<Tensor*> &inputs, const std::vector<Tensor*> &outputs) {
    //MNN_PRINT("cuda convSingleInput onExecute in, inputsize:%d %d\n", (int)inputs.size(), workspace_size_);
    MNN_ASSERT(inputs.size() == 1);
    MNN_ASSERT(outputs.size() == 1);

    auto runtime = static_cast<CUDABackend*>(backend())->getCUDARuntime();
    const void *input_addr = (const void*)inputs[0]->deviceId();
    const void *filter_addr = mResource->mFilter;
    const void *bias_addr = mResource->mBias;

    void *output_addr = (void*)outputs[0]->deviceId();
    void *workspace_addr = nullptr;
    if (workspace_size_ != 0) {
        workspace_addr = mWorkSpace;
    }

    const float alpha = 1;
    const float beta = 0;

    if(use_pad_) {
        std::vector<int> in_shape = {mIOInfo.ib, mIOInfo.ic, mIOInfo.ih, mIOInfo.iw};

        int size = in_shape[0] * in_shape[1] * (in_shape[2]+pad_top_+pad_bottom_) * (in_shape[3]+pad_left_+pad_right_);
        int block_num = runtime->blocks_num(size);
        int threads_num = runtime->threads_num();

        Pad<<<block_num, threads_num>>>(size, (float*)input_addr, in_shape[2], in_shape[3],
            in_shape[2]+pad_top_+pad_bottom_, in_shape[3]+pad_left_+pad_right_, pad_top_, pad_left_, 0.0, (float*)mPadPtr);

        cudnn_check(hipdnnConvolutionForward(cudnn_handle_, &alpha, padded_desc_, mPadPtr, mResource->filter_desc_, filter_addr, mResource->conv_desc_,
            conv_algorithm_, workspace_addr, workspace_size_, &beta, output_desc_, output_addr));
    }
    else {
        cudnn_check(hipdnnConvolutionForward(cudnn_handle_, &alpha, input_desc_, input_addr, mResource->filter_desc_, filter_addr, mResource->conv_desc_,
            conv_algorithm_, workspace_addr, workspace_size_, &beta, output_desc_, output_addr));
    }

    if(mResource->use_bias_) {
        cudnn_check(hipdnnAddTensor(cudnn_handle_, &alpha, mResource->bias_desc_, bias_addr, &alpha, output_desc_, output_addr));
    }
    if(mResource->use_relu_ || mResource->use_relu6_) {
        cudnn_check(hipdnnActivationForward(cudnn_handle_, mResource->act_desc_, &alpha, output_desc_, output_addr, &beta, output_desc_, output_addr));
    }
    
    return NO_ERROR;
}

class CUDAConvolutionCreator : public CUDABackend::Creator {
public:
    virtual Execution* onCreate(const std::vector<Tensor*>& inputs, const std::vector<Tensor*>& outputs, 
            const MNN::Op* op, Backend* backend) const override {
        if (nullptr != op->main_as_Convolution2D()->quanParameter()) {
            auto quan = op->main_as_Convolution2D()->quanParameter();
            if (1 == quan->type() || 2 == quan->type()) {
                if (quan->has_scaleInt()) {
                    // Don't support IDST-int8 because of error
                    return nullptr;
                }
            }
        }
        return new ConvSingleInputExecution(backend, op);
    }
};

CUDACreatorRegister<CUDAConvolutionCreator> __ConvExecution(OpType_Convolution);

}// namespace CUDA
}// namespace MNN
